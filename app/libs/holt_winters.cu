#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void holt_winters_smoothing_kernel(float *values, float *projections, int num_values, int period, float *seasonals) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num_values && idx >= 2 * period) {
        float alpha = 0.5;
        float beta = 0.5;
        float gamma = 0.5;
        float s = values[idx - 2 * period];
        float b = values[idx - 2 * period + 1] - values[idx - 2 * period];
        for (int i = 0; i < period; i++) {
            seasonals[i] = values[idx - 2 * period + i] - s;
        }
        for (int i = idx - 2 * period + period; i <= idx; i++) {
            float value = values[i];
            float last_s = s;
            float last_b = b;
            s = alpha * (value - seasonals[i % period]) + (1 - alpha) * (last_s + last_b);
            b = beta * (s - last_s) + (1 - beta) * last_b;
            seasonals[i % period] = gamma * (value - s) + (1 - gamma) * seasonals[i % period];
        }
        projections[idx] = s + b + seasonals[idx % period];
    }
}



extern "C" {
    void holt_winters_smoothing(float *values, float *projections, int num_values, int period) {
        float *d_values, *d_projections, *d_seasonals;
        hipError_t err;

        err = hipMalloc(&d_values, num_values * sizeof(float));
        if (err != hipSuccess) {
            fprintf(stderr, "Failed to allocate device memory for values: %s\n", hipGetErrorString(err));
            return;
        }

        err = hipMemcpy(d_values, values, num_values * sizeof(float), hipMemcpyHostToDevice);
        if (err != hipSuccess) {
            fprintf(stderr, "Failed to copy values to device: %s\n", hipGetErrorString(err));
            hipFree(d_values);
            return;
        }

        err = hipMalloc(&d_projections, num_values * sizeof(float));
        if (err != hipSuccess) {
            fprintf(stderr, "Failed to allocate device memory for projections: %s\n", hipGetErrorString(err));
            hipFree(d_values);
            return;
        }

        err = hipMalloc(&d_seasonals, period * sizeof(float));
        if (err != hipSuccess) {
            fprintf(stderr, "Failed to allocate device memory for seasonals: %s\n", hipGetErrorString(err));
            hipFree(d_values);
            hipFree(d_projections);
            return;
        }

        int threadsPerBlock = 256;
        int blocksPerGrid = (num_values + threadsPerBlock - 1) / threadsPerBlock;

        holt_winters_smoothing_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_values, d_projections, num_values, period, d_seasonals);
        err = hipGetLastError();
        if (err != hipSuccess) {
            fprintf(stderr, "Failed to launch holt_winters_smoothing_kernel: %s\n", hipGetErrorString(err));
            hipFree(d_values);
            hipFree(d_projections);
            hipFree(d_seasonals);
            return;
        }
    err = hipMemcpy(projections, d_projections, num_values * sizeof(float), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to copy projections to host: %s\n", hipGetErrorString(err));
        hipFree(d_values);
        hipFree(d_projections);
        hipFree(d_seasonals);
        return;
    }

    hipFree(d_values);
    hipFree(d_projections);
    hipFree(d_seasonals);
 }
}