#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

__global__ void interpolation_kernel(float *indices, float *valores, float *result_multivariate, float *result_gaussian, float *result_polynomial, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n - 1) {
        float x = indices[idx];
        float x1 = indices[idx];
        float x2 = indices[idx + 1];
        float y1 = valores[idx];
        float y2 = valores[idx + 1];

        result_multivariate[idx] = (y1 + (x - x1) * (y2 - y1) / (x2 - x1));
        result_gaussian[idx] = (y1 * exp(-0.5 * pow((x - x1) / 1.0, 2)) / (1.0 * sqrt(2 * M_PI)) + y2 * exp(-0.5 * pow((x - x2) / 1.0, 2)) / (1.0 * sqrt(2 * M_PI)));
        result_polynomial[idx] = (y1 + (y2 - y1) * (x - x1) / (x2 - x1));
    }
}

extern "C" {
    void run_interpolation_kernel(float *indices, float *valores, float *result_multivariate, float *result_gaussian, float *result_polynomial, int n) {
        float *d_indices, *d_valores, *d_result_multivariate, *d_result_gaussian, *d_result_polynomial;

        hipMalloc(&d_indices, n * sizeof(float));
        hipMalloc(&d_valores, n * sizeof(float));
        hipMalloc(&d_result_multivariate, n * sizeof(float));
        hipMalloc(&d_result_gaussian, n * sizeof(float));
        hipMalloc(&d_result_polynomial, n * sizeof(float));

        hipMemcpy(d_indices, indices, n * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_valores, valores, n * sizeof(float), hipMemcpyHostToDevice);

        int blockSize = 256;
        int numBlocks = (n + blockSize - 1) / blockSize;
        interpolation_kernel<<<numBlocks, blockSize>>>(d_indices, d_valores, d_result_multivariate, d_result_gaussian, d_result_polynomial, n);

        hipMemcpy(result_multivariate, d_result_multivariate, n * sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(result_gaussian, d_result_gaussian, n * sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(result_polynomial, d_result_polynomial, n * sizeof(float), hipMemcpyDeviceToHost);

        hipFree(d_indices);
        hipFree(d_valores);
        hipFree(d_result_multivariate);
        hipFree(d_result_gaussian);
        hipFree(d_result_polynomial);
    }
}
