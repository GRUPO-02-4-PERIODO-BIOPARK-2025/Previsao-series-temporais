#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void moving_average_kernel(float *values, float *averages, int num_values, int period) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num_values) {
        float sum = 0.0;
        int count = 0;
        for (int i = idx; i < idx + period && i < num_values; i++) {
            sum += values[i];
            count++;
        }
        if (count > 0) {
            averages[idx] = sum / count;
        }
    }
}

extern "C" {
    void moving_average(float *values, float *averages, int num_values, int period) {
        float *d_values, *d_averages;
        hipError_t err;

        err = hipMalloc(&d_values, num_values * sizeof(float));
        if (err != hipSuccess) {
            printf("Erro ao alocar memória na GPU para valores: %s\n", hipGetErrorString(err));
            return;
        }

        err = hipMemcpy(d_values, values, num_values * sizeof(float), hipMemcpyHostToDevice);
        if (err != hipSuccess) {
            printf("Erro ao copiar valores para a GPU: %s\n", hipGetErrorString(err));
            return;
        }

        err = hipMalloc(&d_averages, num_values * sizeof(float));
        if (err != hipSuccess) {
            printf("Erro ao alocar memória na GPU para médias: %s\n", hipGetErrorString(err));
            return;
        }

        int threadsPerBlock = 256;
        int blocksPerGrid = (num_values + threadsPerBlock - 1) / threadsPerBlock;

        printf("Executando o kernel com %d blocos de %d threads cada\n", blocksPerGrid, threadsPerBlock);
        moving_average_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_values, d_averages, num_values, period);
        err = hipGetLastError();
        if (err != hipSuccess) {
            printf("Erro ao executar o kernel: %s\n", hipGetErrorString(err));
            return;
        }

        err = hipMemcpy(averages, d_averages, num_values * sizeof(float), hipMemcpyDeviceToHost);
        if (err != hipSuccess) {
            printf("Erro ao copiar médias de volta para o host: %s\n", hipGetErrorString(err));
            return;
        }

        hipFree(d_values);
        hipFree(d_averages);
    }
}
