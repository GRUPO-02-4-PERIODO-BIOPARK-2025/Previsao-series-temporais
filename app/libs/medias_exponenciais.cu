#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void double_exponential_smoothing(float *values, float **projections, int num_values, int *periods, int num_periods) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num_values) {
        for (int p = 0; p < num_periods; p++) {
            int period = periods[p];
            if (idx >= period) {
                float alpha = 2.0 / (period + 1);
                float beta = alpha / 2;
                float s1 = values[idx - period];
                float s2 = s1;
                for (int i = idx - period + 1; i <= idx; i++) {
                    s1 = alpha * values[i] + (1 - alpha) * s1;
                    s2 = beta * s1 + (1 - beta) * s2;
                }
                projections[p][idx] = 2 * s1 - s2;
            }
        }
    }
}

int main() {
    int num_values = 100; 
    float *values; 
    hipMalloc(&values, num_values * sizeof(float));
    // Copia pra GPU

    int periods[] = {3, 4, 5, 6, 7, 12, 30};
    int num_periods = sizeof(periods) / sizeof(periods[0]);

    float **projections;
    projections = (float **)malloc(num_periods * sizeof(float *));
    for (int i = 0; i < num_periods; i++) {
        hipMalloc(&projections[i], num_values * sizeof(float));
    }

    int *d_periods;
    hipMalloc(&d_periods, num_periods * sizeof(int));
    hipMemcpy(d_periods, periods, num_periods * sizeof(int), hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (num_values + threadsPerBlock - 1) / threadsPerBlock;

    double_exponential_smoothing<<<blocksPerGrid, threadsPerBlock>>>(values, projections, num_values, d_periods, num_periods);

    hipFree(values);
    hipFree(d_periods);
    for (int i = 0; i < num_periods; i++) {
        hipFree(projections[i]);
    }
    free(projections);

    return 0;
}

