#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>

// Função para interpolação gaussiana em 2D
__device__ float gaussian_interpolation_2d(float x, float y, float x1, float y1, float x2, float y2, float f11, float f21, float f12, float f22) {
    float sigma = 1.0; 
    float gauss_x1 = exp(-0.5 * pow((x - x1) / sigma, 2)) / (sigma * sqrt(2 * M_PI));
    float gauss_x2 = exp(-0.5 * pow((x - x2) / sigma, 2)) / (sigma * sqrt(2 * M_PI));
    float gauss_y1 = exp(-0.5 * pow((y - y1) / sigma, 2)) / (sigma * sqrt(2 * M_PI));
    float gauss_y2 = exp(-0.5 * pow((y - y2) / sigma, 2)) / (sigma * sqrt(2 * M_PI));

    return f11 * gauss_x1 * gauss_y1 + f21 * gauss_x2 * gauss_y1 + f12 * gauss_x1 * gauss_y2 + f22 * gauss_x2 * gauss_y2;
}

// Kernel CUDA para realizar a interpolação gaussiana em 2D
__global__ void gaussian_interpolation_2d_kernel(float *input, float *output, int rows, int cols) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < rows && col < cols && input[row * cols + col] == -1) { // Supondo que -1 represente uma string
        // Encontre os quatro pontos mais próximos para interpolação
        int x1 = max(0, col - 1);
        int x2 = min(cols - 1, col + 1);
        int y1 = max(0, row - 1);
        int y2 = min(rows - 1, row + 1);

        float f11 = input[y1 * cols + x1];
        float f21 = input[y1 * cols + x2];
        float f12 = input[y2 * cols + x1];
        float f22 = input[y2 * cols + x2];

        output[row * cols + col] = gaussian_interpolation_2d(col, row, x1, y1, x2, y2, f11, f21, f12, f22);
    } else if (row < rows && col < cols) {
        output[row * cols + col] = input[row * cols + col];
    }
}

void interpolate_2d(float *input, float *output, int rows, int cols) {
    float *d_input, *d_output;

    hipMalloc(&d_input, rows * cols * sizeof(float));
    hipMalloc(&d_output, rows * cols * sizeof(float));

    hipMemcpy(d_input, input, rows * cols * sizeof(float), hipMemcpyHostToDevice);

    dim3 blockSize(16, 16);
    dim3 numBlocks((cols + blockSize.x - 1) / blockSize.x, (rows + blockSize.y - 1) / blockSize.y);
    gaussian_interpolation_2d_kernel<<<numBlocks, blockSize>>>(d_input, d_output, rows, cols);

    hipMemcpy(output, d_output, rows * cols * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);
}

int main() {
    // Exemplo
    int rows = 5;
    int cols = 5;
    float input[] = {
        0, 1, 2, 3, -1,
        2, 4, 5, 8, 2,
        -1, -1, 2, 1, 1,
        2, 2, 3, 5, 8,
        -1, -1, -1, -1, -1
    };
    float output[rows * cols];

    interpolate_2d(input, output, rows, cols);

    printf("Interpolated Matrix:\n");
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            printf("%f ", output[i * cols + j]);
        }
        printf("\n");
    }

    return 0;
}

